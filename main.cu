#include "hip/hip_runtime.h"
// Standard libs
#include <iostream>
#include <chrono>

// GL libs
#include <GL/glut.h>

// CUDA libs
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

// My libs
// Wrapper define
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		// Print message
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) 
			<< " at " << file 
			<< ":" << line 
			<< " '" << func << "' \n" 
			<< hipGetErrorString(result) << "\n";
		// Reset GPU
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}

#include "fpsCounter.h"

#include "vec3.h"
#include "ray.h"
#include "scene.h"
#include "rayHittable.h"
#include "sphere.h"
#include "doubleUtils.h"
#include "camera.h"
#include "material.h"

static const int SAMPLES_PER_PIXEL = 50;
static const int MAX_SCATTER_DEPTH = 20;

#define BACKGROUND_START_GRADIENT_COLOR vec3(0.5, 0.7, 1.0)
#define BACKGROUND_END_GRADIENT_COLOR vec3(1.0, 1.0, 1.0)

const int screenWidth = 960;
const int screenHeight = 480;

__device__ camera dCam;

std::chrono::steady_clock::time_point start, end;

scene** currentScene;
scene** dSceneAll;
scene** dSceneDiffuse;
scene** dSceneMetalic;
scene** dSceneGlass;
scene** dSceneBig;
vec3* fb;
hiprandState* globalState;

int tx = 8;
int ty = 8;
dim3 blocks(screenWidth / tx + 1, screenHeight / ty + 1);
dim3 threads(tx, ty);

__device__ vec3 color(const ray& r, scene** dScene, hiprandState localState)
{
	ray currentRay = r;
	vec3 currentAttenuation = vec3(1.0, 1.0, 1.0);

	// NOTE: Recursion blows up GPU stack, so instead I use iterative recursion
	for (int i = 0; i < MAX_SCATTER_DEPTH; i++) 
	{
		hitInfo hit;
		if ((*dScene)->hit(currentRay, 0.001f, 1000, hit)) 
		{
			ray scattered;
			vec3 attenuation;
			if (hit.matPtr->scatter(currentRay, hit, attenuation, scattered, localState)) 
			{
				currentAttenuation *= attenuation;
				currentRay = scattered;
			}
			else 
			{
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else 
		{
			vec3 unit_direction = unit_vector(currentRay.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * BACKGROUND_END_GRADIENT_COLOR + t * BACKGROUND_START_GRADIENT_COLOR;
			return currentAttenuation * c;
		}
	}
	// Over depth limit
	return vec3(0.0, 0.0, 0.0);
}

__global__ void createCamera(int width, int height)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		dCam = camera(vec3(0, 0, 1), vec3(0,0,-1), vec3(0,1,0), 45.0, width/height);
	}
}

__global__ void updateCameraPosition(vec3 translation)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		dCam.updatePosition(translation);
	}
}

__global__ void setupRNG(hiprandState* globalState, int seed, int screenWidth)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int id = j * screenWidth + i;
	hiprand_init(seed, id, 0, &globalState[id]);
}

__global__ void createScenes(scene** dSceneAll, scene** dSceneDiffuse, scene** dSceneMetalic, scene** dSceneGlass, scene** dSceneBig)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		// All scene
		rayHittable** objects = (rayHittable**)malloc(8 * sizeof(rayHittable*));
		*(objects) = new sphere(vec3(0, 0, -2), 0.5, new lambertian(vec3(0.8, 0.2, 0.2)));
		*(objects + 1) = new sphere(vec3(-2, -1, -5), 1, new lambertian(vec3(0.0, 0.8, 0.8)));
		*(objects + 2) = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.0, 0.8, 0.0)));
		*(objects + 3) = new sphere(vec3(1.5, 0.5, -2), 0.5, new lambertian(vec3(1, 1, 1)));
		*(objects + 4) = new sphere(vec3(-1.5, 0.5, -2), 0.5, new lambertian(vec3(0, 0, 0)));
		*(objects + 5) = new sphere(vec3(1.5, 0, -3), 0.5, new metal(vec3(1, 1, 1), 1));
		*(objects + 6) = new sphere(vec3(-2, 0, -2), 0.5, new metal(vec3(1, 0, 0), 0.5));
		*(objects + 7) = new sphere(vec3(-0.5, 0, -1.5), 0.25, new dielectric(0.9));
		*dSceneAll = new scene(objects, 8);

		// Diffuse scene
		objects = (rayHittable**)malloc(5 * sizeof(rayHittable*));
		*(objects) = new sphere(vec3(0, 0, -1), 0.5, new lambertian(vec3(0.8, 0.2, 0.2)));
		*(objects + 1) = new sphere(vec3(1, 0, -1), 0.5, new lambertian(vec3(0.0, 0.8, 0.8)));
		*(objects + 2) = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.0, 0.8, 0.0)));
		*(objects + 3) = new sphere(vec3(-1, 0, -1), 0.5, new lambertian(vec3(1, 1, 1)));
		*(objects + 4) = new sphere(vec3(-2, 1, 0), 0.5, new lambertian(vec3(0, 0, 1)));
		*dSceneDiffuse = new scene(objects, 5);

		// Metalic scene
		objects = (rayHittable**)malloc(5 * sizeof(rayHittable*));
		*(objects) = new sphere(vec3(0, 0, -1), 0.5, new metal(vec3(1, 1, 1), 1));
		*(objects + 1) = new sphere(vec3(1, 0, -1), 0.5, new metal(vec3(0.5, 0, 0.5), 0.5));
		*(objects + 2) = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.0, 0.8, 0.0)));
		*(objects + 3) = new sphere(vec3(-1, 0, -1), 0.5, new lambertian(vec3(1, 1, 1)));
		*(objects + 4) = new sphere(vec3(-2, 1, 0), 0.5, new metal(vec3(1, 1, 1), 1));
		*dSceneMetalic = new scene(objects, 5);

		// Glass scene
		objects = (rayHittable**)malloc(6 * sizeof(rayHittable*));
		*(objects) = new sphere(vec3(0, 0, -1), 0.5, new dielectric(0.9));
		*(objects + 1) = new sphere(vec3(1, 0, -1), 0.5, new dielectric(0.1));
		*(objects + 2) = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.0, 0.8, 0.0)));
		*(objects + 3) = new sphere(vec3(-1, 0, -1), 0.5, new lambertian(vec3(1, 1, 1)));
		*(objects + 4) = new sphere(vec3(-2, 1, 0), 0.5, new dielectric(5));
		*(objects + 5) = new sphere(vec3(0, 0, -3), 0.5, new lambertian(vec3(0, 0, 1)));
		*dSceneGlass = new scene(objects, 6);

		// Big scene
		objects = (rayHittable**)malloc(300 * sizeof(rayHittable*));
		for (int i = 0; i < 10; i++)
		{
			for (int j = 0; j < 10; j++)
			{
				*(objects + i * 10 + j) = new sphere(vec3((i-4)/2.0, (j - 5)/2.0, -4), 0.1, new lambertian(vec3(i / 5.0, j / 5.0, 0.5)));
				*(objects + i * 10 + j + 100) = new sphere(vec3((i - 4) / 2.0+0.25, (j - 4) / 2.0, -5), 0.2, new metal(vec3((i+j)/100.0, 0.2, 0.5), (10-i + j)/100.0));
				*(objects + i * 10 + j + 200) = new sphere(vec3((i - 4) / 2.0+0.25, (j - 4) +0.5, -6), 0.1, new lambertian(vec3(i / 5.0, j / 5.0, 0.5)));
			}
		}
		*dSceneBig = new scene(objects, 300);
	}
}

__global__ void clearFb(vec3* fb, int maxX)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int pixelIndex = j * maxX + i;
	fb[pixelIndex] = vec3(0.0, 0.0, 0.0);
}

__global__ void render(vec3* fb, int maxX, int maxY, scene** dScene, hiprandState* globalState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	
	int pixelIndex = j * maxX + i;

	for (int p = 0; p < SAMPLES_PER_PIXEL; p++)	// TODO: Maybe divide this across threads
	{
		float u = (float(i) + hiprand_uniform(&globalState[pixelIndex])) / float(maxX);
		float v = (float(j) + hiprand_uniform(&globalState[pixelIndex])) / float(maxY);
		ray r = dCam.getRay(u, v);
#ifndef USE_GAMMA_CORRECTION
		fb[pixelIndex] += color(r, dScene, globalState[pixelIndex]);
#else
		vec3 clr = color(r, dScene, globalState[pixelIndex]);
		clr.v3sqrt();
		fb[pixelIndex] += clr;
#endif
	}
	fb[pixelIndex] /= SAMPLES_PER_PIXEL;
}

void draw()
{
	glBegin(GL_POINTS);

	// Draw out texture
	glPointSize(1.0);
	for (int y = 0; y < screenHeight; y++)
	{
		for (int x = 0; x < screenWidth; x++)
		{
			int ind = y * screenWidth + x;

			glColor3f(fb[ind].x(), fb[ind].y(), fb[ind].z());
			glVertex2f(x, y);
		}
	}
	glEnd();
}

void displayCallback()
{
	glClear(GL_COLOR_BUFFER_BIT);
	draw();
	glFlush();
}

void cleanup()
{
	checkCudaErrors(hipFree(fb));
}

void rerender()
{
	start = std::chrono::high_resolution_clock::now();
	clearFb << <blocks, threads >> > (fb, screenWidth);
	render << <blocks, threads >> > (fb, screenWidth, screenHeight, currentScene, globalState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Render kernel time:\t\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;
	lastRenderTime = (end - start) / std::chrono::milliseconds(1);
}

void glutHandleKeyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 'w':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition <<<1, 1 >>> (vec3(0, 0, -1));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed W" << std::endl;
		rerender();
		break;
	case 's':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition << <1, 1 >> > (vec3(0, 0, 1));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed S" << std::endl;
		rerender();
		break;
	case 'a':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition << <1, 1 >> > (vec3(-1, 0, 0));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed A" << std::endl;
		rerender();
		break;
	case 'd':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition << <1, 1 >> > (vec3(1, 0, 0));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed D" << std::endl;
		rerender();
		break;
	case 'r':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition << <1, 1 >> > (vec3(0, 1, 0));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed R" << std::endl;
		rerender();
		break;
	case 'f':
		start = std::chrono::high_resolution_clock::now();
		updateCameraPosition << <1, 1 >> > (vec3(0, -1, 0));
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		std::cout << "Pressed F" << std::endl;
		rerender();
		break;
	case '1':
		currentScene = dSceneAll;
		std::cout << "Changing to scene 1 (all)" << std::endl;
		rerender();
		break;
	case '2':
		currentScene = dSceneDiffuse;
		std::cout << "Changing to scene 2 (diffuse)" << std::endl;
		rerender();
		break;
	case '3':
		currentScene = dSceneMetalic;
		std::cout << "Changing to scene 3 (metal)" << std::endl;
		rerender();
		break;
	case '4':
		currentScene = dSceneGlass;
		std::cout << "Changing to scene 4 (glass)" << std::endl;
		rerender();
		break;
	case '5':
		currentScene = dSceneBig;
		std::cout << "Changing to scene 5 (big)" << std::endl;
		rerender();
		break;
	}

}

void initGL(int argc, char **args)
{
	glutInit(&argc, args);
	glutInitDisplayMode(GLUT_RGBA);
	glutInitWindowSize(screenWidth, screenHeight);
	glutInitWindowPosition(10, 10);
	glutCreateWindow("RayTracer");

	glMatrixMode(GL_PROJECTION);
	glOrtho(0, screenWidth, 0, screenHeight, -1, 1);
	glMatrixMode(GL_MODELVIEW);

	glClearColor(1.0, 0.0, 1.0, 0.0);

	glutDisplayFunc(displayCallback);
	glutKeyboardFunc(glutHandleKeyboard);
	glutTimerFunc((unsigned int)(1000.0 / TARGET_FPS), refreshFrameCallback, NULL);
	glutTimerFunc(FPS_DISPLAY_REFRESH_TIME, displayFPSCountCallback, NULL);
}

void writeStartInfo()
{
	std::cout << "\tQuality info:" << std::endl
		<< "\t\tSamples per pixel: " << SAMPLES_PER_PIXEL << std::endl
		<< "\t\tScatter depth: " << MAX_SCATTER_DEPTH << std::endl
		<< "\t\tGamma correction: " <<
#ifdef DIFFUSE_HALF_SPHERE
		"Yes"
#else
		"No"
#endif
		<< std::endl
		<< "\t\tDiffuse half sphere: " <<
#ifdef USE_GAMMA_CORRECTION
		"Yes"
#else
		"No"
#endif
		<< std::endl;
}

int main(int argc, char** args)
{
	int numPixels = screenWidth * screenHeight;

	writeStartInfo();

	// Init GL
	initGL(argc, args);
	std::cout << "GL initialized" << std::endl;

	// CUDA mallocs
	start = std::chrono::high_resolution_clock::now();
	checkCudaErrors(hipMallocManaged((void**)&dSceneAll, sizeof(scene)));
	checkCudaErrors(hipMallocManaged((void**)&dSceneDiffuse, sizeof(scene)));
	checkCudaErrors(hipMallocManaged((void**)&dSceneMetalic, sizeof(scene)));
	checkCudaErrors(hipMallocManaged((void**)&dSceneGlass, sizeof(scene)));
	checkCudaErrors(hipMallocManaged((void**)&dSceneBig, sizeof(scene)));
	checkCudaErrors(hipMallocManaged(&fb, numPixels * sizeof(vec3)));
	checkCudaErrors(hipMallocManaged(&globalState, numPixels * sizeof(hiprandState)));
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Cuda mallocs time:\t\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Create scene kernel
	start = std::chrono::high_resolution_clock::now();
	createScenes <<<1, 1>>> (dSceneAll, dSceneDiffuse, dSceneMetalic, dSceneGlass, dSceneBig);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Create scene kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;
	
	// Create camera kernel
	start = std::chrono::high_resolution_clock::now();
	createCamera << <1, 1 >> > (screenWidth, screenHeight);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Create camera kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Setup RNG kernel
	start = std::chrono::high_resolution_clock::now();
	setupRNG << <blocks, threads >> > (globalState, time(NULL), screenWidth);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "RNG states init kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Render kernel
	currentScene = dSceneDiffuse;
	std::cout << "Changing to scene 2 (diffuse)" << std::endl;
	rerender();

	glutMainLoop();

	return 0;
}