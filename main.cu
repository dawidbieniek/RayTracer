#include "hip/hip_runtime.h"
// Standard libs
#include <iostream>

// GL libs
#include <GL/glut.h>

// CUDA libs
#include "hip/hip_runtime.h"
#include ""

// My libs
//#include "cudaHelpers.h"
// TMP
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
// !TMP

int screenWidth = 640;
int screenHeight = 480;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void render(float* fb, int maxX, int maxY)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= maxX) || (j >= maxY)) return;

	int pixelIndex = j * maxX * 3 + i * 3;
	fb[pixelIndex + 0] = float(i) / maxX;	// R
	fb[pixelIndex + 1] = float(j) / maxY;	// G
	fb[pixelIndex + 2] = 0.2;				// B
}

float* fb;

void draw()
{
	glBegin(GL_POINTS);

	glPointSize(1.0);
	for (int y = 0; y < screenHeight; y++)
	{
		for (int x = 0; x < screenWidth; x++)
		{
			int ind = 3*(y * screenWidth + x);

			glColor3f(fb[ind], fb[ind + 1], fb[ind + 2]);
			//std::cout << fb[ind] << "  " << fb[ind + 1] << "  " << fb[ind + 2] << std::endl;
			glVertex2f(x, y);
		}
	}

	glEnd();
}


void display()
{
	glClear(GL_COLOR_BUFFER_BIT);
	draw();
	glFlush();
}

void cleanup()
{
	checkCudaErrors(hipFree(fb));
}

int main(int argc, char** argv)
{
	// Example
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Good
	int numPixels = screenWidth * screenHeight;
	size_t fbSize = 3 * static_cast<size_t>(numPixels) * sizeof(float);
	//fb = (float*)malloc(fbSize);
	//for (int y = 50; y < 100; y++)
	//{
	//	for (int x = 0; x < screenWidth; x++)
	//	{
	//		fb[3*(y * screenWidth + x)] = 1.0;
	//		fb[3 * (y * screenWidth + x) + 1] = 0.5;
	//		fb[3 * (y * screenWidth + x) + 2] = 1.0;
	//	}
	//}
	checkCudaErrors(hipMallocManaged(&fb, fbSize));

	int tx = 8;
	int ty = 8;
	dim3 blocks(screenWidth / tx + 1, screenHeight / ty + 1);
	dim3 threads(tx, ty);

	render <<<blocks, threads >>> (fb, screenWidth, screenHeight);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// GL Init
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA);
	glutInitWindowSize(screenWidth, screenHeight);
	glutInitWindowPosition(10, 10);
	glutCreateWindow("OKNO");

	glMatrixMode(GL_PROJECTION);
	glOrtho(0, screenWidth, 0, screenHeight, -1, 1);
	glMatrixMode(GL_MODELVIEW);

	glClearColor(1.0, 0.0, 1.0, 0.0);
	
	glutDisplayFunc(display);
	glutMainLoop();

	//cleanup();
	//std::cout << "End of program" << std::endl;

	//// Add vectors in parallel.
	//hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "addWithCuda failed!");
	//	return 1;
	//}

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//	c[0], c[1], c[2], c[3], c[4]);

	//// hipDeviceReset must be called before exiting in order for profiling and
	//// tracing tools such as Nsight and Visual Profiler to show complete traces.
	//cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	return 1;
	//}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}