#include "hip/hip_runtime.h"
// Standard libs
#include <iostream>
#include <chrono>

// GL libs
#include <GL/glut.h>

// CUDA libs
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

// My libs
// Wrapper define
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line)
{
	if (result)
	{
		// Print message
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) 
			<< " at " << file 
			<< ":" << line 
			<< " '" << func << "' \n" 
			<< hipGetErrorString(result) << "\n";
		// Reset GPU
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}

#include "vec3.h"
#include "ray.h"
#include "scene.h"
#include "rayHittable.h"
#include "sphere.h"
#include "doubleUtils.h"
#include "camera.h"
#include "material.h"

static const int TARGET_FPS = 60;
static const unsigned int FPS_DISPLAY_REFRESH_TIME = 500;
static const int SAMPLES_PER_PIXEL = 50;
static const int MAX_DIFFUSE_DEPTH = 20;
static const int SCENE_ELEMENTS = 8;

#define BACKGROUND_START_GRADIENT_COLOR vec3(0.5, 0.7, 1.0)
#define BACKGROUND_END_GRADIENT_COLOR vec3(1.0, 1.0, 1.0)

const int screenWidth = 960;
const int screenHeight = 480;

__device__ camera dCam;

// Returns length of ray from origin to hit point. -1 if not hit
__device__ double sphereHitPoint(const vec3& center, float radius, const ray& r)
{
	vec3 oc = r.origin() - center;
	auto a = r.direction().lengthSquared();
	auto half_b = dot(oc, r.direction());
	auto c = oc.lengthSquared() - radius * radius;
	auto discriminant = half_b * half_b - a * c;

	if (discriminant < 0) return -1.0;
	return (-half_b - sqrt(discriminant)) / a;
}

__device__ vec3 color(const ray& r, scene** dScene, hiprandState localState)
{
	ray currentRay = r;
	vec3 currentAttenuation = vec3(1.0, 1.0, 1.0);

	// NOTE: Recursion blows up GPU stack, so instead I use iterative recursion
	for (int i = 0; i < MAX_DIFFUSE_DEPTH; i++) 
	{
		hitInfo hit;
		if ((*dScene)->hit(currentRay, 0.001f, INFINITY, hit)) 
		{
			ray scattered;
			vec3 attenuation;
			if (hit.matPtr->scatter(currentRay, hit, attenuation, scattered, localState)) 
			{
				currentAttenuation *= attenuation;
				currentRay = scattered;
			}
			else 
			{
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else 
		{
			vec3 unit_direction = unit_vector(currentRay.direction());
			float t = 0.5f * (unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t) * BACKGROUND_END_GRADIENT_COLOR + t * BACKGROUND_START_GRADIENT_COLOR;
			return currentAttenuation * c;
		}
	}
	// Over depth limit
	return vec3(0.0, 0.0, 0.0);
}

__global__ void createCamera(int width, int height)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		dCam = camera(vec3(0, 0, 1), vec3(0,0,-1), vec3(0,1,0), 45.0, width/height);
	}
}

__global__ void setupRNG(hiprandState* globalState, int seed, int screenWidth)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	int id = j * screenWidth + i;
	hiprand_init(seed, id, 0, &globalState[id]);
}

__global__ void createScene(rayHittable** dObjects, scene** dScene)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		*(dObjects) = new sphere(vec3(0, 0, -2), 0.5, new lambertian(vec3(0.8, 0.2, 0.2)));
		*(dObjects + 1) = new sphere(vec3(-2, -1, -5), 1, new lambertian(vec3(0.0, 0.8, 0.8)));
		*(dObjects + 2) = new sphere(vec3(0, -100.5, -1), 100, new lambertian(vec3(0.0, 0.8, 0.0)));
		*(dObjects + 3) = new sphere(vec3(1.5, 0.5, -2), 0.5, new lambertian(vec3(1, 1, 1)));
		*(dObjects + 4) = new sphere(vec3(-1.5, 0.5, -2), 0.5, new lambertian(vec3(0, 0, 0)));
		*(dObjects + 5) = new sphere(vec3(1.5, 0, -3), 0.5, new metal(vec3(1, 1, 1), 1));
		*(dObjects + 6) = new sphere(vec3(-2, 0, -2), 0.5, new metal(vec3(1, 0, 0), 0.5));
		*(dObjects + 7) = new sphere(vec3(-0.5, 0, -1.5), 0.25, new dielectric(0.9));
		*dScene = new scene(dObjects, SCENE_ELEMENTS);
	}
}

__global__ void render(vec3* fb, int maxX, int maxY, scene** dScene, hiprandState* globalState)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if ((i >= maxX) || (j >= maxY)) return;
	
	int pixelIndex = j * maxX + i;
	hiprandState localState = globalState[pixelIndex];
	for (int p = 0; p < SAMPLES_PER_PIXEL; p++)	// TODO: Maybe divide this across threads
	{
		float u = (float(i) + hiprand_uniform(&localState)) / float(maxX);
		float v = (float(j) + hiprand_uniform(&localState)) / float(maxY);
		ray r = dCam.getRay(u, v);
#ifndef USE_GAMMA_CORRECTION
		fb[pixelIndex] += color(r, dScene, localState);
#else
		vec3 clr = color(r, dScene, localState);
		clr.v3sqrt();
		fb[pixelIndex] += clr;
#endif
	}
	fb[pixelIndex] /= SAMPLES_PER_PIXEL;
}

vec3* fb;

int lastFrameTime = 0;
int deltaTimes[TARGET_FPS];
int deltaTimesIndex = 0;

void refreshFrameCallback(int value)
{
	if (glutGetWindow()) 
	{
		int currentTime = glutGet(GLUT_ELAPSED_TIME);
		int deltaTime = currentTime - lastFrameTime;
		lastFrameTime = currentTime;
		deltaTimes[deltaTimesIndex++] = deltaTime;
		if (deltaTimesIndex >= TARGET_FPS)
			deltaTimesIndex = 0;

		// Refresh window
		glutPostRedisplay();
		// Refresh callback
		glutTimerFunc((unsigned int)(1000.0 / TARGET_FPS), refreshFrameCallback, NULL);
	}
}

void displayFPSCountCallback(int value)
{
	if (glutGetWindow())
	{
		float fpsCount = 0;
		int i;
		for (i = 0; i < TARGET_FPS; i++)
		{
			if (deltaTimes[i] == 0)
				break;
			fpsCount += deltaTimes[i];
		}

		fpsCount  = 1000.0 / (fpsCount / i + 1);

		char titleBuffer[16];
		sprintf(titleBuffer, "FPS: %3.1f", fpsCount);
		glutSetWindowTitle(titleBuffer);

		// Refresh callback
		glutTimerFunc(FPS_DISPLAY_REFRESH_TIME, displayFPSCountCallback, NULL);
	}
}

void draw()
{
	glBegin(GL_POINTS);

	// Draw out texture
	glPointSize(1.0);
	for (int y = 0; y < screenHeight; y++)
	{
		for (int x = 0; x < screenWidth; x++)
		{
			int ind = y * screenWidth + x;

			glColor3f(fb[ind].x(), fb[ind].y(), fb[ind].z());
			glVertex2f(x, y);
		}
	}
	glEnd();
}

void displayCallback()
{
	glClear(GL_COLOR_BUFFER_BIT);
	draw();
	glFlush();
}

void cleanup()
{
	checkCudaErrors(hipFree(fb));
}

void initGL(int argc, char **args)
{
	glutInit(&argc, args);
	glutInitDisplayMode(GLUT_RGBA);
	glutInitWindowSize(screenWidth, screenHeight);
	glutInitWindowPosition(10, 10);
	glutCreateWindow("OKNO");

	glMatrixMode(GL_PROJECTION);
	glOrtho(0, screenWidth, 0, screenHeight, -1, 1);
	glMatrixMode(GL_MODELVIEW);

	glClearColor(1.0, 0.0, 1.0, 0.0);

	glutDisplayFunc(displayCallback);
	glutTimerFunc((unsigned int)(1000.0 / TARGET_FPS), refreshFrameCallback, NULL);
	glutTimerFunc(FPS_DISPLAY_REFRESH_TIME, displayFPSCountCallback, NULL);
}

int main(int argc, char** args)
{
	std::chrono::steady_clock::time_point start, end;

	int numPixels = screenWidth * screenHeight;

	hiprandState* globalState;

	scene** dScene;
	rayHittable** dObjects;

	int tx = 8;
	int ty = 8;
	dim3 blocks(screenWidth / tx + 1, screenHeight / ty + 1);
	dim3 threads(tx, ty);

	// Init GL
	initGL(argc, args);

	// CUDA mallocs
	start = std::chrono::high_resolution_clock::now();
	checkCudaErrors(hipMallocManaged((void**)&dObjects, SCENE_ELEMENTS * sizeof(rayHittable*)));
	checkCudaErrors(hipMallocManaged((void**)&dScene, sizeof(scene)));
	checkCudaErrors(hipMallocManaged(&fb, numPixels * sizeof(vec3)));
	checkCudaErrors(hipMallocManaged(&globalState, numPixels * sizeof(hiprandState)));
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Cuda mallocs time:\t\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Create scene kernel
	start = std::chrono::high_resolution_clock::now();
	createScene <<<1, 1>>> (dObjects, dScene);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Create scene kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;
	
	// Create camera kernel
	start = std::chrono::high_resolution_clock::now();
	createCamera << <1, 1 >> > (screenWidth, screenHeight);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Create camera kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Setup RNG kernel
	start = std::chrono::high_resolution_clock::now();
	setupRNG << <blocks, threads >> > (globalState, time(NULL), screenWidth);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "RNG states init kernel time:\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	// Render kernel
	start = std::chrono::high_resolution_clock::now();
	render <<<blocks, threads >>> (fb, screenWidth, screenHeight, dScene, globalState);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	end = std::chrono::high_resolution_clock::now();
	std::cout << "Render kernel time:\t\t\t" << (end - start) / std::chrono::milliseconds(1) << "\tms" << std::endl;

	glutMainLoop();

	return 0;
}